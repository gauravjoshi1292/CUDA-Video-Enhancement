#include "hip/hip_runtime.h"
#include "opencv2/opencv.hpp"
#pragma comment (lib , "opencv_core244d.lib")
#pragma comment (lib ,"opencv_highgui244d.lib")
#pragma comment(lib , "opencv_imgproc244d.lib")
#include <cv.h>
#include <highgui.h>
#include"CImg.h"
#include"SOIL.h"
#include<conio.h>
#include<stdio.h>
#include<iostream>
#include<math.h>
#include<time.h>
#include<hip/hip_runtime.h>
#include<GL/glut.h>
#include<GL/GLU.h>
#include<GL/GL.h>
#include<Windows.h>

using namespace cimg_library;
using namespace cv;
using namespace std;

CImg<unsigned char> imageup(1024,1024,1,3);

int FLAG=0;
char *str3=new char[50];


//function for capturing frames either continuously or by skipping them by a number n
IplImage* skipNFrames(CvCapture* capture, int n)
{
    for(int i = 0; i < n; ++i)
    {
        if(cvQueryFrame(capture) == NULL)
        {
            return NULL;
        }
    }

    return cvQueryFrame(capture);
}

//function for calculating I11 on a GPU device
__device__ int I11(int i,int j,int *arr,int width)
{
	return (arr[(i - 2)+ (j - 2)*width] + arr[(i + 2) + (j + 2)*width] - 2*arr[i + (j)*width]);
}

//function for calculating I22 on a GPU device
__device__ int I22(int i,int j,int *arr,int width)
{
	return (arr[(i - 2)+ (j + 2)*width]+arr[(i + 2)+ (j - 2)*width]+arr[i + (j)*width]);
}

//function for finding absolute value, executed onn a GPU
__device__ int mod(int i)
{
	if (i<0)
		return ((-1)*i);
	else
		return i;
}

__constant__ int T; //the thresshold value
__constant__ char del; //the delta value

//function for calculating energy executed on a GPU device, calculates energy for a pixel value
__device__ float energycalc(int i,int j,int *arr,int w1,int w2,int w3,int w4,int width)
{ 
	float u,uc,ue;
 	uc=w1*(mod((I11(i,j,arr,width) - I11(i+1,j+1,arr,width))) +
                     mod(I22(i,j,arr,width) - I22(i+1,j+1,arr,width))) +
				w2*(mod(I11(i,j,arr,width) - I11(i+1,j-1,arr,width)) +
					mod(I22(i,j,arr,width) - I22(i+1,j-1,arr,width)))+
				w3*(mod(I11(i,j,arr,width) - I11(i-1,j+1,arr,width)) +
					mod(I22(i,j,arr,width) - I22(i-1,j+1,arr,width))) +
				w4*(mod(I11(i,j,arr,width) - I11(i-1,j-1,arr,width)) +
					mod(I22(i,j,arr,width) - I22(i-1,j-1,arr,width)));
			  ue=-1*(mod(I11(i,j,arr,width))+mod(I22(i,j,arr,width)));
			  u=5*uc-2*ue;
return u;

}

//function for finding pixel values for (2i+1, 2j+1), a global function
__global__ void FastInterpolation1(int *red,int *green,int *blue,int width, int height)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x+1;
	int j = (blockIdx.y * blockDim.y) + threadIdx.y+1;
	float secd1,secd2,secd3,secd4,secd5,secd6;
	
			secd1= red[(2*i-2)+width*(2*j + 2)] + red[(2*i)+width* (2*j)]+red[(2*i + 2)+width* (2*j - 2)] - 3*red[(2*i)+width* (2*j + 2)]-3*red[(2*i + 2)+width* 2*j] +red[(2*i)+width*( 2*j + 4)] + red[(2*i + 2)+width*( 2*j + 2)] + red[(2*i + 4)+width* 2*j];
			secd2=red[(2*i)+width*( 2*j-2)] + red[(2*i + 2)+width* 2*j]+red[(2*i + 4)+width*( 2*j + 2)] - 3*red[(2*i)+width* (2*j)] -3*red[(2*i + 2)+width* (2*j + 2)]+red[(2*i- 2)+width* (2*j)] + red[(2*i)+width* (2*j + 2)] + red[(2*i + 2)+width* (2*j + 4)];
			if(secd1<=secd2)
				red[(2*i+1)+width*(2*j+1)]=(red[(2*i)+width*(2*j)]+red[(2*i+2)+width*(2*j+2)])/2;
			else
				red[(2*i+1)+width*(2*j+1)]=(red[(2*i+2)+width*2*j]+red[2*i+width*(2*j+2)])/2;
			
			secd3= green[(2*i-2)+width*(2*j + 2)] + green[(2*i)+width* (2*j)]+green[(2*i + 2)+width* (2*j - 2)] - 3*green[(2*i)+width* (2*j + 2)]-3*green[(2*i + 2)+width* 2*j] +green[(2*i)+width*(2*j + 4)] + green[(2*i + 2)+width* (2*j + 2)] + green[(2*i + 4)+width* 2*j];
			secd4=green[(2*i)+width* (2*j-2)] + green [(2*i + 2)+width* (2*j)]+green [(2*i + 4)+width* (2*j + 2)] - 3*green [(2*i)+width* (2*j)] -3*green [(2*i + 2)+width* (2*j + 2)]+green [(2*i- 2)+width* (2*j)] + green [(2*i)+width* (2*j + 2)] + green[(2*i + 2)+width* (2*j + 4)];
			if(secd3<=secd4)
				green[(2*i+1)+width*(2*j+1)]=(green[(2*i)+width*(2*j)]+green[(2*i+2)+width*(2*j+2)])/2;
			else
				green[(2*i+1)+width*(2*j+1)]=(green[(2*i+2)+width*2*j]+green[2*i+width*(2*j+2)])/2;
			
			secd5= blue[(2*i-2)+width*(2*j + 2)] + blue[(2*i)+width* (2*j)]+blue[(2*i + 2)+width* (2*j - 2)] - 3*blue[(2*i)+width* (2*j + 2)]-3*blue[(2*i + 2)+width* (2*j)] +blue[(2*i)+ width*(2*j + 4)] + blue[(2*i + 2)+width* (2*j + 2)] + blue[(2*i + 4)+width* 2*j];
			secd6=blue[(2*i)+width* (2*j-2)] + blue[(2*i + 2)+width* 2*j]+blue[(2*i + 4)+width* (2*j + 2)] - 3*blue[(2*i)+width* (2*j)] -3*blue[(2*i + 2)+width* (2*j + 2)]+blue[(2*i- 2)+width* 2*j] + blue[(2*i)+width* (2*j + 2)] + blue[(2*i + 2)+width* (2*j + 4)];
			if(secd5<=secd6)
				blue[(2*i+1)+width*(2*j+1)]=(blue [(2*i)+width*(2*j)]+blue [(2*i+2)+width*(2*j+2)])/2;
			else
				blue[(2*i+1)+width*(2*j+1)]=(blue [(2*i+2)+width*(2*j)]+blue [(2*i)+width*(2*j+2)])/2;
}

//function for finding pixel values (2i,2j+1)
__global__ void FastInterpolation2(int *red,int *green,int *blue,int width, int height)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x+1;
	int j = (blockIdx.y * blockDim.y) + threadIdx.y+1;
	float secd1,secd2;
	
			secd1= red[(2*i-3)+width*(2*j + 2)] + red[(2*i-1)+width* (2*j)]+red[(2*i + 1)+width* (2*j - 2)] - 3*red[(2*i-1)+width* (2*j + 2)]-3*red[(2*i +1)+width* 2*j] +red[(2*i-1)+width*( 2*j + 4)] + red[(2*i + 1)+width*( 2*j + 2)] + red[(2*i + 3)+width* 2*j];
			secd2=red[(2*i-1)+width*( 2*j-2)] + red[(2*i + 1)+width* 2*j]+red[(2*i + 3)+width*( 2*j + 2)] - 3*red[(2*i-1)+width* (2*j)] -3*red[(2*i + 1)+width* (2*j + 2)]+red[(2*i- 3)+width* (2*j)] + red[(2*i-1)+width* (2*j + 2)] + red[(2*i + 1)+width* (2*j + 4)];
			if(secd1<=secd2)
				red[(2*i)+width*(2*j+1)]=(red[(2*i)+width*(2*j)]+red[(2*i+2)+width*(2*j+2)])/2;
			else
				red[(2*i)+width*(2*j+1)]=(red[(2*i+2)+width*2*j]+red[2*i+width*(2*j+2)])/2;
			
			secd1= green[(2*i-3)+width*(2*j + 2)] + green[(2*i-1)+width* (2*j)]+green[(2*i + 1)+width* (2*j - 2)] - 3*green[(2*i-1)+width* (2*j + 2)]-3*green[(2*i + 1)+width* 2*j] +green[(2*i-1)+width*(2*j + 4)] + green[(2*i + 1)+width* (2*j + 2)] + green[(2*i + 2)+width* 2*j];
			secd2=green[(2*i-1)+width* (2*j-2)] + green [(2*i + 1)+width* (2*j)]+green [(2*i + 2)+width* (2*j + 2)] - 3*green [(2*i-1)+width* (2*j)] -3*green [(2*i + 1)+width* (2*j + 1)]+green [(2*i- 3)+width* (2*j)] + green [(2*i-1)+width* (2*j + 2)] + green[(2*i + 1)+width* (2*j + 4)];
			if(secd1<=secd2)
				green[2*i+width*(2*j+1)]=(green[2*i+width*2*j]+green[2*i+2+width*(2*j+2)])/2;
			else
				green[2*i+width*(2*j+1)]=(green[2*i+2+width*2*j]+green[2*i+width*(2*j+2)])/2;
			
			secd1= blue[2*i-3+width*(2*j + 2)] + blue[2*i-1+width* 2*j]+blue[2*i-1 +width* (2*j - 2)] - 3*blue[2*i-1+width* (2*j + 2)]-3*blue[2*i+1 +width* 2*j] +blue[2*i-1+ (2*j + 4)*width] + blue[2*i+1 + 2+width* (2*j + 2)] + blue[2*i + 3+width* 2*j];
			secd2=blue[2*i-1+width* (2*j-2)] + blue[2*i + 1+width* 2*j]+blue[2*i + 3+width* (2*j + 1)] - 3*blue[2*i+width* 2*j] -3*blue[2*i-1 + 2+width* (2*j + 2)]+blue[2*i- 3+width* 2*j] + blue[2*i-1+width* (2*j + 2)] + blue[2*i+1 + 2+width* (2*j + 4)];
			if(secd1<=secd2)
				blue[2*i+width*(2*j+1)]=(blue [2*i+width*2*j]+blue [2*i+2+width*(2*j+2)])/2;
			else
				blue[2*i+width*(2*j+1)]=(blue [2*i+2+width*2*j]+blue [2*i+width*(2*j+2)])/2;
}

//function for finding pixel values (2i+1,2j)
__global__ void FastInterpolation3(int *red,int *green,int *blue,int width, int height)
{
    int i = (blockIdx.x * blockDim.x) + threadIdx.x+1;
	int j = (blockIdx.y * blockDim.y) + threadIdx.y+1;
	float secd1,secd2;
	
			secd1= red[(2*i-2)+width*(2*j + 1)] + red[(2*i)+width* (2*j-1)]+red[(2*i + 2)+width* (2*j - 3)] - 3*red[(2*i)+width* (2*j + 1)]-3*red[(2*i+2)+width* (2*j-1)] +red[(2*i)+width*( 2*j + 3)] + red[(2*i + 2)+width*( 2*j + 1)] + red[(2*i + 4)+width* (2*j-1)];
			secd2=red[(2*i)+width*( 2*j-3)] + red[(2*i + 2)+width* (2*j-1)]+red[(2*i + 4)+width*( 2*j + 1)] - 3*red[(2*i)+width* (2*j-1)] -3*red[(2*i+2)+width* (2*j +1)]+red[(2*i- 2)+width* (2*j-1)] + red[(2*i)+width* (2*j + 1)] + red[(2*i + 2)+width* (2*j + 3)];
			if(secd1<=secd2)
				red[(2*i+1)+width*(2*j)]=(red[(2*i+1)+width*(2*j-1)]+red[(2*i+3)+width*(2*j+1)])/2;
			else
				red[(2*i+1)+width*(2*j)]=(red[(2*i+3)+width*(2*j-1)]+red[(2*i+1)+width*(2*j+1)])/2;
			
			secd1= blue[(2*i-2)+width*(2*j + 1)] + blue[(2*i)+width* (2*j-1)]+blue[(2*i + 2)+width* (2*j - 3)] - 3*blue[(2*i)+width* (2*j + 1)]-3*blue[(2*i+2)+width* (2*j-1)] +blue[(2*i)+width*( 2*j + 3)] + blue[(2*i + 2)+width*( 2*j + 1)] + blue[(2*i + 4)+width* (2*j-1)];
			secd2=blue[(2*i)+width*( 2*j-3)] + blue[(2*i + 2)+width* (2*j-1)]+blue[(2*i + 4)+width*( 2*j + 1)] - 3*blue[(2*i)+width* (2*j-1)] -3*blue[(2*i+2)+width* (2*j +1)]+blue[(2*i- 2)+width* (2*j-1)] + blue[(2*i)+width* (2*j + 1)] + blue[(2*i + 2)+width* (2*j + 3)];
			if(secd1<=secd2)
				blue[(2*i+1)+width*(2*j)]=(blue[(2*i+1)+width*(2*j-1)]+blue[(2*i+3)+width*(2*j+1)])/2;
			else
				blue[(2*i+1)+width*(2*j)]=(blue[(2*i+3)+width*(2*j-1)]+blue[(2*i+1)+width*(2*j+1)])/2;
			
			secd1= green[(2*i-2)+width*(2*j + 1)] + green[(2*i)+width* (2*j-1)]+green[(2*i + 2)+width* (2*j - 3)] - 3*green[(2*i)+width* (2*j + 1)]-3*green[(2*i+2)+width* (2*j-1)] +green[(2*i)+width*( 2*j + 3)] + green[(2*i + 2)+width*( 2*j + 1)] + green[(2*i + 4)+width* (2*j-1)];
			secd2=green[(2*i)+width*( 2*j-3)] + green[(2*i + 2)+width* (2*j-1)]+green[(2*i + 4)+width*( 2*j + 1)] - 3*green[(2*i)+width* (2*j-1)] -3*green[(2*i+2)+width* (2*j +1)]+green[(2*i- 2)+width* (2*j-1)] + green[(2*i)+width* (2*j + 1)] + green[(2*i + 2)+width* (2*j + 3)];
			if(secd1<=secd2)
				green[(2*i+1)+width*(2*j)]=(green[(2*i+1)+width*(2*j-1)]+green[(2*i+3)+width*(2*j+1)])/2;
			else
				green[(2*i+1)+width*(2*j)]=(green[(2*i+3)+width*(2*j-1)]+green[(2*i+1)+width*(2*j+1)])/2;
}

//function for applying iterative correction to (2i+1,2j+1)
__global__ void Iterations1(int *red,int *green,int *blue,int width, int height)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x+1;
	int j = (blockIdx.y * blockDim.y) + threadIdx.y+1;
	int save;
	float i1,i2,i3,i4,min,uplus,uminus;
	int flag,w1=0,w2=0,w3=0,w4=0;
	for(int l=1;l<1;l++)
	{
	  		  i1= -0.5*(red[(2*i+2)+(2*j+2)*width]-red[(2*i+1)+(2*j+1)*width]);
			  i2= -0.5*(red[(2*i+2)+(2*j)*width]-red[(2*i+1)+(2*j+1)*width]);
			  i3= -0.5*(red[(2*i)+(2*j+2)*width]-red[(2*i+1)+(2*j+1)*width]);
			  i4= -0.5*(red[(2*i)+(2*j)*width]-red[(2*i+1)+(2*j+1)*width]);
			  if (i1<T)
				  w1=1;
			  else if (i1>=T)
				  w1=0;
			  if (i2<T)
				  w2=1;
			  else if (i2>=T)
				  w2=0;
			  if (i3<T)
				  w3=1;
			  else if (i3>=T)
				  w3=0;
			  if (i4<T)
				  w4=1;
			  else if (i4>=T)
				  w4=0;
			 min=energycalc(2*i+1,2*j+1,red,w1,w2,w3,w4,width);flag=0;
			 syncthreads();
			 save=red[(2*i+1)+(2*j+1)*width];
			 red[(2*i+1)+(2*j+1)*width]=save+del;
			 syncthreads();
			 uplus=energycalc(2*i+1,2*j+1,red,w1,w2,w3,w4,width);
			 if(uplus<min)
			 {min=uplus;flag=1;}
			 red [(2*i+1)+(2*j+1)*width]=save-del;
			 syncthreads();
			 uminus=energycalc(2*i+1,2*j+1,red,w1,w2,w3,w4,width);
			 if(uminus<min)
			 {min=uminus;flag=2;}

			 switch(flag)
			 {
			 case 0:
				 {red[(2*i+1)+(2*j+1)*width]=save;
				 break;}
			 case 1:
				 {red[(2*i+1)+(2*j+1)*width]=save+del;
				 break;}
			 case 2:
				 {red[(2*i+1)+(2*j+1)*width]=save-del;
				 break;}
			 }
			  i1= -0.5*(blue[(2*i+2)+(2*j+2)*width]-blue[(2*i+1)+(2*j+1)*width]);
			  i2= -0.5*(blue[(2*i+2)+(2*j)*width]-blue[(2*i+1)+(2*j+1)*width]);
			  i3= -0.5*(blue[(2*i)+(2*j+2)*width]-blue[(2*i+1)+(2*j+1)*width]);
			  i4= -0.5*(blue[(2*i)+(2*j)*width]-blue[(2*i+1)+(2*j+1)*width]);
			  if (i1<T)
				  w1=1;
			  else if (i1>=T)
				  w1=0;
			  if (i2<T)
				  w2=1;
			  else if (i2>=T)
				  w2=0;
			  if (i3<T)
				  w3=1;
			  else if (i3>=T)
				  w3=0;
			  if (i4<T)
				  w4=1;
			  else if (i4>=T)
				  w4=0;
			  
		     min=energycalc(2*i+1,2*j+1,blue,w1,w2,w3,w4,width);flag=0;
			 syncthreads();
			 save=blue[(2*i+1)+(2*j+1)*width];
			 blue[(2*i+1)+(2*j+1)*width]=save+del;
			 syncthreads();
			 uplus=energycalc(2*i+1,2*j+1,blue,w1,w2,w3,w4,width);
			 if(uplus<min)
			 {min=uplus;flag=1;}
			 blue[(2*i+1)+(2*j+1)*width]=save-del;
			 syncthreads();
			 uminus=energycalc(2*i+1,2*j+1,blue,w1,w2,w3,w4,width);
			 if(uminus<min)
			 {min=uminus;flag=2;}

			 switch(flag)
			 {
			 case 0:
				 {blue[(2*i+1)+(2*j+1)*width]=save;
				 break;}
			 case 1:
				 {blue[(2*i+1)+(2*j+1)*width]=save+del;
				 break;}
			 case 2:
				 {blue[(2*i+1)+(2*j+1)*width]=save-del;
				 break;}
			 }
			 
			  i1= -0.5*(green[(2*i+2)+(2*j+2)*width]-green[(2*i+1)+(2*j+1)*width]);
			  i2= -0.5*(green[(2*i+2)+(2*j)*width]-green[(2*i+1)+(2*j+1)*width]);
			  i3= -0.5*(green[(2*i)+(2*j+2)*width]-green[(2*i+1)+(2*j+1)*width]);
			  i4= -0.5*(green[(2*i)+(2*j)*width]-green[(2*i+1)+(2*j+1)*width]);
			  if (i1<T)
				  w1=1;
			  else if (i1>=T)
				  w1=0;
			  if (i2<T)
				  w2=1;
			  else if (i2>=T)
				  w2=0;
			  if (i3<T)
				  w3=1;
			  else if (i3>=T)
				  w3=0;
			  if (i4<T)
				  w4=1;
			  else if (i4>=T)
				  w4=0;
			  
		     min=energycalc(2*i+1,2*j+1,green,w1,w2,w3,w4,width);flag=0;
			 syncthreads();
			 save=green[(2*i+1)+(2*j+1)*width];
			 green[(2*i+1)+(2*j+1)*width]=save+del;
			 syncthreads();
			 uplus=energycalc(2*i+1,2*j+1,green,w1,w2,w3,w4,width);
			 if(uplus<min)
			 {min=uplus;flag=1;}
			 green[(2*i+1)+(2*j+1)*width]=save-del;
			 syncthreads();
			 uminus=energycalc(2*i+1,2*j+1,green,w1,w2,w3,w4,width);
			 if(uminus<min)
			 {min=uminus;flag=2;}

			 switch(flag)
			 {
			 case 0:
				 {green[(2*i+1)+(2*j+1)*width]=save;
				 break;}
			 case 1:
				 {green[(2*i+1)+(2*j+1)*width]=save+del;
				 break;}
			 case 2:
				 {green[(2*i+1)+(2*j+1)*width]=save-del;
				 break;}
			 }
		__syncthreads(); 
       }
}

//function for applying iterative correction to (2i,2j+1)
__global__ void Iterations2(int *red,int *green,int *blue,int width, int height)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x+1;
	int j = (blockIdx.y * blockDim.y) + threadIdx.y+1;
	char save;
	float i1,i2,i3,i4,min,uplus,uminus;int flag,w1=0,w2=0,w3=0,w4=0;
	for(int l=1;l<1;l++)
	{
	  		  i1= -0.5*(red[2*i+1+(2*j+2)*width]-red[2*i+(2*j+1)*width]);
			  i2= -0.5*(red[2*i+1+(2*j)*width]-red[2*i+(2*j+1)*width]);
			  i3= -0.5*(red[2*i-1+(2*j+2)*width]-red[2*i+(2*j+1)*width]);
			  i4= -0.5*(red[2*i-1+(2*j)*width]-red[2*i+(2*j+1)*width]);
			  if (i1<T)
				  w1=1;
			  else if (i1>=T)
				  w1=0;
			  if (i2<T)
				  w2=1;
			  else if (i2>=T)
				  w2=0;
			  if (i3<T)
				  w3=1;
			  else if (i3>=T)
				  w3=0;
			  if (i4<T)
				  w4=1;
			  else if (i4>=T)
				  w4=0;
			  
		     min=energycalc(2*i,2*j+1,red,w1,w2,w3,w4,width);flag=0;
			 syncthreads();
			 save=red[2*i+(2*j+1)*width];
			 red[2*i+(2*j+1)*width]=save+del;
			 syncthreads();
			 uplus=energycalc(2*i+1,2*j+1,red,w1,w2,w3,w4,width);
			 if(uplus<min)
			 {min=uplus;flag=1;}
			red[2*i+(2*j+1)*width]=save-del;
			syncthreads(); 
			uminus=energycalc(2*i,2*j+1,red,w1,w2,w3,w4,width);
			 if(uminus<min)
			 {min=uminus;flag=2;}

			 switch(flag)
			 {
			 case 0:
				 {red[2*i+(2*j+1)*width]=save;
				 break;}
			 case 1:
				 {red[2*i+(2*j+1)*width]=save+del;
				 break;}
			 case 2:
				 {red[2*i+(2*j+1)*width]=save-del;
				 break;}
			 }
			  i1= -0.5*(blue[2*i+1+(2*j+2)*width]-blue[2*i+(2*j+1)*width]);
			  i2= -0.5*(blue[2*i+1+(2*j)*width]-blue[2*i+(2*j+1)*width]);
			  i3= -0.5*(blue[2*i-1+(2*j+2)*width]-blue[2*i+(2*j+1)*width]);
			  i4= -0.5*(blue[2*i-1+(2*j)*width]-blue[2*i+(2*j+1)*width]);
			  if (i1<T)
				  w1=1;
			  else if (i1>=T)
				  w1=0;
			  if (i2<T)
				  w2=1;
			  else if (i2>=T)
				  w2=0;
			  if (i3<T)
				  w3=1;
			  else if (i3>=T)
				  w3=0;
			  if (i4<T)
				  w4=1;
			  else if (i4>=T)
				  w4=0;
			  
		     min=energycalc(2*i,2*j+1,blue,w1,w2,w3,w4,width);flag=0;
			 syncthreads();
			 save=blue[2*i+(2*j+1)*width];
			 blue[2*i+(2*j+1)*width]=save+del;
			 syncthreads();
			 uplus=energycalc(2*i,2*j+1,blue,w1,w2,w3,w4,width);
			 if(uplus<min)
			 {min=uplus;flag=1;}
			 blue[2*i+(2*j+1)*width]=save-del;
			 syncthreads();
			 uminus=energycalc(2*i,2*j+1,blue,w1,w2,w3,w4,width);
			 if(uminus<min)
			 {min=uminus;flag=2;}

			 switch(flag)
			 {
			 case 0:
				 {blue[2*i+(2*j+1)*width]=save;
				 break;}
			 case 1:
				 {blue[2*i+(2*j+1)*width]=save+del;
				 break;}
			 case 2:
				 {blue[2*i+(2*j+1)*width]=save-del;
				 break;}
			 }
			 
			  i1= -0.5*(green[2*i+1+(2*j+2)*width]-green[2*i+(2*j+1)*width]);
			  i2= -0.5*(green[2*i+1+(2*j)*width]-green[2*i+(2*j+1)*width]);
			  i3= -0.5*(green[2*i-1+(2*j+2)*width]-green[2*i+(2*j+1)*width]);
			  i4= -0.5*(green[2*i-1+(2*j)*width]-green[2*i+(2*j+1)*width]);
			  if (i1<T)
				  w1=1;
			  else if (i1>=T)
				  w1=0;
			  if (i2<T)
				  w2=1;
			  else if (i2>=T)
				  w2=0;
			  if (i3<T)
				  w3=1;
			  else if (i3>=T)
				  w3=0;
			  if (i4<T)
				  w4=1;
			  else if (i4>=T)
				  w4=0;
			  
		     min=energycalc(2*i,2*j+1,green,w1,w2,w3,w4,width);flag=0;
			 syncthreads();
			 save=green[2*i+(2*j+1)*width];
			 green[2*i+(2*j+1)*width]=save+del;
			 syncthreads();
			 uplus=energycalc(2*i,2*j+1,green,w1,w2,w3,w4,width);
			 if(uplus<min)
			 {min=uplus;flag=1;}
			 green[2*i+(2*j+1)*width]=save-del;
			 syncthreads();
			 uminus=energycalc(2*i,2*j+1,green,w1,w2,w3,w4,width);
			 if(uminus<min)
			 {min=uminus;flag=2;}

			 switch(flag)
			 {
			 case 0:
				 {green[2*i+(2*j+1)*width]=save;
				 break;}
			 case 1:
				 {green[2*i+(2*j+1)*width]=save+del;
				 break;}
			 case 2:
				 {green[2*i+(2*j+1)*width]=save-del;
				 break;}
			 }
		__syncthreads(); 
       }
}

//function for applying iterative correction to (2i+1,2j)
__global__ void Iterations3(int *red,int *green,int *blue,int width, int height)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x+1;
	int j = (blockIdx.y * blockDim.y) + threadIdx.y+1;
	int save;
	float i1,i2,i3,i4,min,uplus,uminus;int flag,w1=0,w2=0,w3=0,w4=0;
	for(int l=1;l<1;l++)
	{
	  		  i1= -0.5*(red[2*i+2+(2*j+1)*width]-red[2*i+1+(2*j)*width]);
			  i2= -0.5*(red[2*i+2+(2*j-1)*width]-red[2*i+1+(2*j)*width]);
			  i3= -0.5*(red[2*i+(2*j+1)*width]-red[2*i+1+(2*j)*width]);
			  i4= -0.5*(red[2*i+(2*j-1)*width]-red[2*i+1+(2*j)*width]);
			  if (i1<T)
				  w1=1;
			  else if (i1>=T)
				  w1=0;
			  if (i2<T)
				  w2=1;
			  else if (i2>=T)
				  w2=0;
			  if (i3<T)
				  w3=1;
			  else if (i3>=T)
				  w3=0;
			  if (i4<T)
				  w4=1;
			  else if (i4>=T)
				  w4=0;
			  
		     min=energycalc(2*i+1,2*j,red,w1,w2,w3,w4,width);flag=0;
			 syncthreads();
			 save=red[2*i+1+(2*j)*width];
			 red[2*i+1+(2*j)*width]=save+del;
			 syncthreads();
			 uplus=energycalc(2*i+1,2*j,red,w1,w2,w3,w4,width);
			 if(uplus<min)
			 {min=uplus;flag=1;}
			 red[2*i+1+(2*j)*width]=save-del;
			 syncthreads();
			 uminus=energycalc(2*i+1,2*j,red,w1,w2,w3,w4,width);
			 if(uminus<min)
			 {min=uminus;flag=2;}

			 switch(flag)
			 {
			 case 0:
				 {red[2*i+1+(2*j)*width]=save;
				 break;}
			 case 1:
				 {red[2*i+1+(2*j)*width]=save+del;
				 break;}
			 case 2:
				 {red[2*i+1+(2*j)*width]=save-del;
				 break;}
			 }
			 i1= -0.5*(blue[2*i+2+(2*j+1)*width]-blue[2*i+1+(2*j)*width]);
			  i2= -0.5*(blue[2*i+2+(2*j-1)*width]-blue[2*i+1+(2*j)*width]);
			  i3= -0.5*(blue[2*i+(2*j+1)*width]-blue[2*i+1+(2*j)*width]);
			  i4= -0.5*(blue[2*i+(2*j-1)*width]-blue[2*i+1+(2*j)*width]);
			  if (i1<T)
				  w1=1;
			  else if (i1>=T)
				  w1=0;
			  if (i2<T)
				  w2=1;
			  else if (i2>=T)
				  w2=0;
			  if (i3<T)
				  w3=1;
			  else if (i3>=T)
				  w3=0;
			  if (i4<T)
				  w4=1;
			  else if (i4>=T)
				  w4=0;
			  
		     min=energycalc(2*i+1,2*j,blue,w1,w2,w3,w4,width);flag=0;
			 syncthreads();
			 save=blue[2*i+1+(2*j)*width];
			 blue[2*i+1+(2*j)*width]=save+del;
			 syncthreads();
			 uplus=energycalc(2*i+1,2*j,blue,w1,w2,w3,w4,width);
			 if(uplus<min)
			 {min=uplus;flag=1;}
			 blue[2*i+1+(2*j)*width]=save-del;
			 syncthreads();
			 uminus=energycalc(2*i+1,2*j,blue,w1,w2,w3,w4,width);
			 if(uminus<min)
			 {min=uminus;flag=2;}

			 switch(flag)
			 {
			 case 0:
				 {blue[2*i+1+(2*j)*width]=save;
				 break;}
			 case 1:
				 {blue[2*i+1+(2*j)*width]=save+del;
				 break;}
			 case 2:
				 {blue[2*i+1+(2*j)*width]=save-del;
				 break;}
			 }
			 
			  i1= -0.5*(green[2*i+2+(2*j+1)*width]-green[2*i+1+(2*j)*width]);
			  i2= -0.5*(green[2*i+2+(2*j-1)*width]-green[2*i+1+(2*j)*width]);
			  i3= -0.5*(green[2*i+(2*j+1)*width]-green[2*i+1+(2*j)*width]);
			  i4= -0.5*(green[2*i+(2*j-1)*width]-green[2*i+1+(2*j)*width]);
			  if (i1<T)
				  w1=1;
			  else if (i1>=T)
				  w1=0;
			  if (i2<T)
				  w2=1;
			  else if (i2>=T)
				  w2=0;
			  if (i3<T)
				  w3=1;
			  else if (i3>=T)
				  w3=0;
			  if (i4<T)
				  w4=1;
			  else if (i4>=T)
				  w4=0;
			  
		     min=energycalc(2*i+1,2*j,green,w1,w2,w3,w4,width);flag=0;
			 syncthreads();
			 save=green[2*i+1+(2*j)*width];
			 green[2*i+1+(2*j)*width]=save+del;
			 syncthreads();
			 uplus=energycalc(2*i+1,2*j,green,w1,w2,w3,w4,width);
			 if(uplus<min)
			 {min=uplus;flag=1;}
			 green[2*i+1+(2*j)*width]=save-del;
			 syncthreads();
			 uminus=energycalc(2*i+1,2*j,green,w1,w2,w3,w4,width);
			 if(uminus<min)
			 {min=uminus;flag=2;}

			 switch(flag)
			 {
			 case 0:
				 {green[2*i+1+(2*j)*width]=save;
				 break;}
			 case 1:
				 {green[2*i+1+(2*j)*width]=save+del;
				 break;}
			 case 2:
				 {green[2*i+1+(2*j)*width]=save-del;
				 break;}
			 }
		__syncthreads(); 
       }
}

void resize(int height, int width) {
    const float ar = (float) width / (float) height;
    glViewport(0, 10, width, height);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();

    glFrustum(-ar, ar, -1.0, 1.0, 2.0, 90.0);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity() ;
}
static void Draw(void)
{   
	GLuint texture;
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    glLoadIdentity();
    glTranslatef(0.0f,0.0f,-3.01f);
    texture = SOIL_load_OGL_texture // load an image file directly as a new OpenGL texture
    (
        str3,
        SOIL_LOAD_AUTO,
        SOIL_CREATE_NEW_ID,
        SOIL_FLAG_MIPMAPS | SOIL_FLAG_INVERT_Y | SOIL_FLAG_NTSC_SAFE_RGB | SOIL_FLAG_COMPRESS_TO_DXT
    );
	glBindTexture(GL_TEXTURE_2D, texture);
    glBegin(GL_QUADS);
    glTexCoord2f(0.0f, 0.0f); glVertex3f(-1.0f, -1.0f,  1.0f);
    glTexCoord2f(1.0f, 0.0f); glVertex3f( 1.0f, -1.0f,  1.0f);
    glTexCoord2f(1.0f, 1.0f); glVertex3f( 1.0f,  1.0f,  1.0f);
    glTexCoord2f(0.0f, 1.0f); glVertex3f(-1.0f,  1.0f,  1.0f);
    glEnd();
    glutSwapBuffers();
}
	int main(int argc, char **argv)
	{
	static int count1=0,count2=0;
	clock_t begin, end;
    double time_spent;
	float min,secd1,secd2,u,uplus,uminus;
	int save,flag,x1,x2,x3,max;
	int *red;
	int *green;
	int *blue;
	int *devPtrR;
	int *devPtrG;
	int *devPtrB;
	int th=22;
	int delh=2;
	float elapsedTime;
	int width = 1024, height =1024;
	int loopOverFrames=0;
	dim3 grid(63,63);
	dim3 blockallot(8,8);
	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop ) ;
	//Allocating 2D arrays on GPU for copying data
	hipMalloc((void**)&devPtrR, sizeof(int) * width*height);
	hipMalloc((void**)&devPtrG, sizeof(int) * width*height);
	hipMalloc((void**)&devPtrB, sizeof(int) * width*height);
	hipMemcpyToSymbol(HIP_SYMBOL(T),&th,1*sizeof(int),0,hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(del),&delh,1*sizeof(char),0,hipMemcpyHostToDevice);
	red=(int*)malloc(width*height*sizeof(int));
	green=(int*)malloc(width*height*sizeof(int));
	blue=(int*)malloc(width*height*sizeof(int));
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
    glutInit(&argc, argv);
    glutInitWindowSize(1024,1024);
    glutCreateWindow("COP - CUDA VIDEO ENHANCEMENT");
    glutReshapeFunc(resize);
    glutDisplayFunc(Draw);
	glEnable(GL_TEXTURE_2D);
    glShadeModel(GL_SMOOTH);
    glClearColor(0.0f, 0.0f, 0.0f, 0.5f);
    glClearDepth(1.0f);
    glEnable(GL_DEPTH_TEST);
    glDepthFunc(GL_LEQUAL);
    glHint(GL_PERSPECTIVE_CORRECTION_HINT, GL_NICEST);
	begin=clock();	
	
	//video capturing first part
	CvCapture* capture = cvCaptureFromFile("try.flv");
	IplImage* frame = NULL;
    do
    {
        frame = skipNFrames(capture, 1);
        cvNamedWindow("frame", CV_WINDOW_AUTOSIZE);
        cvShowImage("frame", frame);
        //cvWaitKey(0);
		char *str=new char[50];
		FLAG++;
		sprintf(str,"%d",FLAG);
		strcat(str,"_frame");
		strcat(str,".jpg");
		Mat image=frame;
		imwrite(str,image);
    }while(frame!=NULL );

    cvReleaseCapture(&capture);
    cvDestroyWindow("frame");
    cvReleaseImage(&frame);   
	
	
	for(loopOverFrames=1;loopOverFrames<=10;loopOverFrames++)
	{
		char *str2=new char[50];
		sprintf(str2,"%d",loopOverFrames);
		strcat(str2,"_frame");
		strcat(str2,".jpg");
		//cout<<str2;
		CImg<unsigned char> image(str2);

//Initializing the new upscaled image
	
	for(int j=0;j<512;j++)
    {
	  for(int i=0;i<512;i++)
	  {
		  for(int k=0;k<3;k++)
		  {  
			 
			  switch(k)
				  {
					case 0:
					{
						red[(2*i)+(2*j)*width]=image(i,j,k);
						imageup((2*i),(2*j),k)=red[(2*i)+(2*j)*width];
						break;
					}
				   case 1:
					{
						green[(2*i)+(2*j)*width]=image(i,j,k);
						imageup((2*i),(2*j),k)=green[(2*i)+(2*j)*width];
						break;
					}
					case 2:
					{
						blue[(2*i)+(2*j)*width]=image(i,j,k);
						imageup((2*i),(2*j),k)=blue[(2*i)+(2*j)*width];
						break;
					}
				 }
		  }
	  }
	}
	
    //kernel call for FAST INTERPOLATION
	hipEventRecord( start, 0 );
	hipMemcpy(devPtrR, red, sizeof(int) * width*height, hipMemcpyHostToDevice);
	hipMemcpy(devPtrG, green, sizeof(int) * width*height, hipMemcpyHostToDevice);
	hipMemcpy(devPtrB, blue, sizeof(int) * width*height, hipMemcpyHostToDevice);
	
	FastInterpolation1<<<grid,blockallot>>>(devPtrR,devPtrG,devPtrB,width,height);
	 
	hipMemcpy(red,devPtrR, sizeof(int) * width*height, hipMemcpyDeviceToHost);
	hipMemcpy(green,devPtrG, sizeof(int) * width*height, hipMemcpyDeviceToHost);
	hipMemcpy(blue,devPtrB, sizeof(int) * width*height, hipMemcpyDeviceToHost);
	
	//kernel call for iterating 2i+1,2j+1
	
	/*
	hipMemcpy(devPtrR, red, sizeof(int) * width*height, hipMemcpyHostToDevice);
	hipMemcpy(devPtrG, green, sizeof(int) * width*height, hipMemcpyHostToDevice);
	hipMemcpy(devPtrB, blue, sizeof(int) * width*height, hipMemcpyHostToDevice);
	
	Iterations1<<<grid,blockallot>>>(devPtrR,devPtrG,devPtrB,width,height);
	
	hipMemcpy(red,devPtrR, sizeof(int) * width*height, hipMemcpyDeviceToHost);
	hipMemcpy(green,devPtrG, sizeof(int) * width*height, hipMemcpyDeviceToHost);
	hipMemcpy(blue,devPtrB, sizeof(int) * width*height, hipMemcpyDeviceToHost);
	*/
	
	//STAGE 2 - FILLING the other two
	//filling 2i,2j+1
	hipMemcpy(devPtrR, red, sizeof(int) * width*height, hipMemcpyHostToDevice);
	hipMemcpy(devPtrG, green, sizeof(int) * width*height, hipMemcpyHostToDevice);
	hipMemcpy(devPtrB, blue, sizeof(int) * width*height, hipMemcpyHostToDevice);
	
	FastInterpolation2<<<grid,blockallot>>>(devPtrR,devPtrG,devPtrB,width,height);
	 
	hipMemcpy(red,devPtrR, sizeof(int) * width*height, hipMemcpyDeviceToHost);
	hipMemcpy(green,devPtrG, sizeof(int) * width*height, hipMemcpyDeviceToHost);
	hipMemcpy(blue,devPtrB, sizeof(int) * width*height, hipMemcpyDeviceToHost);
	/*
	hipMemcpy(devPtrR, red, sizeof(int) * width*height, hipMemcpyHostToDevice);
	hipMemcpy(devPtrG, green, sizeof(int) * width*height, hipMemcpyHostToDevice);
	hipMemcpy(devPtrB, blue, sizeof(int) * width*height, hipMemcpyHostToDevice);
	
	Iterations2<<<grid,blockallot>>>(devPtrR,devPtrG,devPtrB,width,height);
	
	hipMemcpy(red,devPtrR, sizeof(int) * width*height, hipMemcpyDeviceToHost);
	hipMemcpy(green,devPtrG, sizeof(int) * width*height, hipMemcpyDeviceToHost);
	hipMemcpy(blue,devPtrB, sizeof(int) * width*height, hipMemcpyDeviceToHost);
	*/
	//kernel call for iterations
	
	//filling 2i+1,2j
	hipMemcpy(devPtrR, red, sizeof(int) * width*height, hipMemcpyHostToDevice);
	hipMemcpy(devPtrG, green, sizeof(int) * width*height, hipMemcpyHostToDevice);
	hipMemcpy(devPtrB, blue, sizeof(int) * width*height, hipMemcpyHostToDevice);
	
	FastInterpolation3<<<grid,blockallot>>>(devPtrR,devPtrG,devPtrB,width,height);
	 
	hipMemcpy(red,devPtrR, sizeof(int) * width*height, hipMemcpyDeviceToHost);
	hipMemcpy(green,devPtrG, sizeof(int) * width*height, hipMemcpyDeviceToHost);
	hipMemcpy(blue,devPtrB, sizeof(int) * width*height, hipMemcpyDeviceToHost);
	
	
	
	/*
	//kernel call for iterations
	hipMemcpy(devPtrR, red, sizeof(int) * width*height, hipMemcpyHostToDevice);
	hipMemcpy(devPtrG, green, sizeof(int) * width*height, hipMemcpyHostToDevice);
	hipMemcpy(devPtrB, blue, sizeof(int) * width*height, hipMemcpyHostToDevice);
	
	Iterations3<<<grid,blockallot>>>(devPtrR,devPtrG,devPtrB,width,height);
	
	hipMemcpy(red,devPtrR, sizeof(int) * width*height, hipMemcpyDeviceToHost);
	hipMemcpy(green,devPtrG, sizeof(int) * width*height, hipMemcpyDeviceToHost);
	hipMemcpy(blue,devPtrB, sizeof(int) * width*height, hipMemcpyDeviceToHost);
	*/
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop ) ;
	hipEventElapsedTime( &elapsedTime,start, stop );
	hipEventDestroy( start );  
	hipEventDestroy( stop );
	printf( "Time to generate: %3.1f ms\n", elapsedTime ); 
	//Iterating for the N times
	
	for(int j=1;j<510;j++)
    {
	  for(int i=1;i<510;i++)
	  {
		  for(int k=0;k<3;k++)
		  {
			switch(k)
				  {
					case 0:
					{
						imageup((2*i+1),(2*j+1),k)=red[(2*i+1)+width*(2*j+1)];
						imageup((2*i+1),(2*j),k)=red[(2*i+1)+width*(2*j)];
						imageup(2*i,2*j+1,k)=red[2*i+(2*j+1)*width];
						break;
					}
				   case 1:
					{
						imageup((2*i+1),(2*j+1),k)=green[(2*i+1)+width*(2*j+1)];
						imageup((2*i+1),(2*j),k)=green[(2*i+1)+width*(2*j)];
						imageup(2*i,2*j+1,k)=green[2*i+(2*j+1)*width];

						break;
					}
					case 2:
					{
						imageup((2*i+1),(2*j+1),k)=blue[(2*i+1)+width*(2*j+1)];
						imageup((2*i+1),(2*j),k)=blue[(2*i+1)+width*(2*j)];
						imageup(2*i,2*j+1,k)=blue[2*i+(2*j+1)*width];
						break;
					}
				 }
			 }
	  }
	 
	}
			
	 for(int j=2;j<1020;j++)
    {
	  for(int i=2;i<1020;i++)
	  {
		  for(int k=0;k<3;k++)
		  { if((imageup(i,j,k)>1.05*imageup(i+1,j,k))&&(imageup(i,j,k)>1.05*imageup(i-1,j,k)))
		           imageup(i,j,k)=(imageup(i-1,j,k)+imageup(i+1,j,k))/2;
		  }
	  }
	 }
	 for(int j=1;j<4;j++)
    {
	  for(int i=0;i<1024;i++)
	  {
		  for(int k=0;k<3;k++)
		  { 
			  if((imageup(i,2*j+1,k)>1.05*imageup(i,2*j,k))&&(imageup(i,2*j+1,k)>1.05*imageup(i,2*j+2,k)))
		           imageup(i,2*j+1,k)=(imageup(i,2*j,k)+imageup(i,2*j+2,k))/2;
		  }
	  }
	 }
	 for(int j=508;j<510;j++)
    {
	  for(int i=0;i<1024;i++)
	  {
		  for(int k=0;k<3;k++)
		  { 
			  if((imageup(i,2*j+1,k)>1.05*imageup(i,2*j,k))&&(imageup(i,2*j+1,k)>1.05*imageup(i,2*j+2,k)))
		           imageup(i,2*j+1,k)=(imageup(i,2*j,k)+imageup(i,2*j+2,k))/2;
		  }
	  }
	 }
	 for(int j=1;j<1023;j++)
    {
	  for(int i=2;i<10;i++)
	  {
		  for(int k=0;k<3;k++)
		  { 
			  if((imageup(i,j,k)>1.05*imageup(i,j-1,k))&&(imageup(i,j,k)>1.05*imageup(i,j+1,k)))
		           imageup(i,j,k)=(imageup(i,j-1,k)+imageup(i,j+1,k))/2;
		  }
	  }
	 }
	  for(int j=1;j<1023;j++)
    {
	  for(int i=1017;i<1020;i++)
	  {
		  for(int k=0;k<3;k++)
		  { 
			  if((imageup(i,j,k)>1.05*imageup(i,j-1,k))&&(imageup(i,j,k)>1.05*imageup(i,j+1,k)))
		           imageup(i,j,k)=(imageup(i,j-1,k)+imageup(i,j+1,k))/2;
		  }
	  }
	 }

	  //filling up pixels at boundary//
	for(int j=0;j<3;j++){
		for(int i=0;i<510;i++){
			for(int k=0;k<3;k++){
				imageup(2*i+1,2*j+1,k)=(imageup(2*i,2*j,k)+imageup(2*i+2,2*j,k)+imageup(2*i+2,2*j+2,k)+imageup(2*i,2*j+2,k))/4;
				imageup(2*i+1,2*j,k)=(imageup(2*i,2*j,k)+imageup(2*i+2,2*j,k)+imageup(2*i+1,2*j+1,k))/3;
			}
		}
	}
	for(int j=0;j<3;j++){
		for(int i=1;i<510;i++){
			for(int k=0;k<3;k++){
				imageup(2*i,2*j+1,k)=(imageup(2*i,2*j,k)+imageup(2*i+1,2*j+1,k)+imageup(2*i,2*j+2,k)+imageup(2*i-1,2*j+1,k))/4;
			}
		}
	}
	for(int j=510;j<511;j++){
		for(int i=0;i<510;i++){
			for(int k=0;k<3;k++){
				imageup(2*i+1,2*j+1,k)=(imageup(2*i,2*j,k)+imageup(2*i+2,2*j,k)+imageup(2*i+2,2*j+2,k)+imageup(2*i,2*j+2,k))/4;
				if(i==0)
					imageup(2*i,2*j+1,k)=(imageup(2*i,2*j,k)+imageup(2*i,2*j+2,k)+imageup(2*i+1,2*j+1,k))/3;
				else
					imageup(2*i,2*j+1,k)=(imageup(2*i,2*j,k)+imageup(2*i,2*j+2,k)+imageup(2*i+1,2*j+1,k)+imageup(2*i-1,2*j+1,k))/4;
				imageup(2*i+1,2*j,k)=(imageup(2*i,2*j,k)+imageup(2*i+2,2*j,k)+imageup(2*i+1,2*j+1,k))/3;
			}
		}
	}
	for(int j=511;j<512;j++){
		for(int i=0;i<510;i++){
			for(int k=0;k<3;k++){
				imageup(2*i+1,2*j,k)=(imageup(2*i,2*j,k)+imageup(2*i+2,2*j,k)+imageup(2*i+1,2*j-1,k))/3;
				imageup(2*i+1,2*j+1,k)=imageup(2*i+1,2*j,k);
				imageup(2*i,2*j+1,k)=imageup(2*i,2*j,k);
			}
		}
	}
	
	for(int j=0;j<510;j++){
		for(int i=0;i<3;i++){
			for(int k=0;k<3;k++){
				imageup(2*i+1,2*j+1,k)=(imageup(2*i,2*j,k)+imageup(2*i+2,2*j,k)+imageup(2*i+2,2*j+2,k)+imageup(2*i,2*j+2,k))/4;
				imageup(2*i,2*j+1,k)=(imageup(2*i,2*j,k)+imageup(2*i,2*j+2,k)+imageup(2*i+1,2*j+1,k))/3;
			}
		}
	}
	for(int j=1;j<510;j++){
		for(int i=0;i<3;i++){
			for(int k=0;k<3;k++){
				imageup(2*i+1,2*j,k)=(imageup(2*i,2*j,k)+imageup(2*i+1,2*j-1,k)+imageup(2*i+2,2*j,k)+imageup(2*i+1,2*j+1,k))/4;
			}
		}
	}
	for(int j=0;j<510;j++){
		for(int i=510;i<511;i++){
			for(int k=0;k<3;k++){
				imageup(2*i+1,2*j+1,k)=(imageup(2*i,2*j,k)+imageup(2*i+2,2*j,k)+imageup(2*i+2,2*j+2,k)+imageup(2*i,2*j+2,k))/4;
				if(j==0)
					imageup(2*i+1,2*j,k)=(imageup(2*i,2*j,k)+imageup(2*i+2,2*j,k)+imageup(2*i+1,2*j+1,k))/3;
				else
					imageup(2*i+1,2*j,k)=(imageup(2*i,2*j,k)+imageup(2*i+2,2*j,k)+imageup(2*i+1,2*j+1,k)+imageup(2*i+1,2*j-1,k))/4;
				imageup(2*i,2*j+1,k)=(imageup(2*i,2*j,k)+imageup(2*i,2*j+2,k)+imageup(2*i+1,2*j+1,k))/3;
			}
		}
	}
	for(int j=0;j<510;j++){
		for(int i=511;i<512;i++){
			for(int k=0;k<3;k++){
				imageup(2*i,2*j+1,k)=(imageup(2*i,2*j,k)+imageup(2*i,2*j+2,k)+imageup(2*i-1,2*j+1,k))/3;
				imageup(2*i+1,2*j+1,k)=imageup(2*i,2*j+1,k);
				imageup(2*i+1,2*j,k)=imageup(2*i,2*j,k);
			}
		}
	}
	
	for(int j=510;j<512;j++){
		for(int i=510;i<512;i++){
			for(int k=0;k<3;k++){
				if(i==510 && j==510){
					imageup(2*i+1,2*j+1,k)=(imageup(2*i,2*j,k)+imageup(2*i,2*j+2,k)+imageup(2*i+2,2*j+2,k)+imageup(2*i+2,2*j,k))/4;
					imageup(2*i,2*j+1,k)=(imageup(2*i,2*j,k)+imageup(2*i,2*j+2,k)+imageup(2*i-1,2*j+1,k)+imageup(2*i+1,2*j+1,k))/4;
					imageup(2*i+1,2*j,k)=(imageup(2*i,2*j,k)+imageup(2*i+2,2*j,k)+imageup(2*i+1,2*j-1,k)+imageup(2*i+1,2*j+1,k))/4;
				}
				if(i==511 && j==510){
					imageup(2*i,2*j+1,k)=(imageup(2*i,2*j,k)+imageup(2*i,2*j+2,k)+imageup(2*i-1,2*j+1,k))/3;
				}
				if(i==510 && j==511){
					imageup(2*i+1,2*j,k)=(imageup(2*i,2*j,k)+imageup(2*i+2,2*j,k)+imageup(2*i+1,2*j-1,k))/3;
				}
				if(i==511){
					imageup(2*i+1,2*j+1,k)=imageup(2*i,2*j+1,k);
					imageup(2*i+1,2*j,k)=imageup(2*i,2*j,k);
				}
				if(j==511){
					imageup(2*i+1,2*j+1,k)=imageup(2*i+1,2*j,k);
					imageup(2*i,2*j+1,k)=imageup(2*i,2*j,k);
				}
			}
		}
	}
		sprintf(str3,"%d",loopOverFrames);
		strcat(str3,"_upscaledFrame");
		strcat(str3,".bmp");
		cout<<str3;
		imageup.save(str3);
		glutMainLoop();
			}
	end=clock();
	 time_spent = (double)(end - begin) / CLOCKS_PER_SEC;
	printf("Time in millisec = %f\n",time_spent);
	getch();
	 return 0;

}

